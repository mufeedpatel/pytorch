
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCStorage.cu"
#else

void THCStorage_(fill)(THCState *state, THCStorage *self, real value)
{
  THCThrustAllocator thrustAlloc(state);
  thrust::device_ptr<real> self_data(THCStorage_(data)(state, self));
  thrust::fill(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
    self_data, self_data+self->size, value);
}

void THCStorage_(resize)(THCState *state, THCStorage *self, ptrdiff_t size)
{
  THCStorage_resize(state, self, size);
}

THC_API int THCStorage_(getDevice)(THCState* state, const THCStorage* storage) {
  return THCStorage_getDevice(state, storage);
}

#endif
